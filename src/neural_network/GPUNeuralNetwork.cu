#include "hip/hip_runtime.h"
//
// Created by volodya-petarda on 3/28/24.
//
#include "GPUNeuralNetwork.h"

GPUNeuralNetwork::GPUNeuralNetwork() {

}

GPUNeuralNetwork::GPUNeuralNetwork(std::vector<int> sizes) {
    this->layers = sizes.size();
    this->sizes = sizes;
    createArrays();
}

void GPUNeuralNetwork::printA() {
    std::cout<<"a:\n";
    for(int i = 0; i < layers; i++){
        float *temp = new float[sizes[i]];
        hipMemcpy(temp, a[i], sizes[i] * sizeof(float), hipMemcpyDeviceToHost);
        for(int j = 0; j < sizes[i]; j++){
            std::cout<<temp[j]<<' ';
        }
        std::cout<<'\n';
        delete[] temp;
    }
}

void GPUNeuralNetwork::printB() {
    std::cout<<"b:\n";
    for(int i = 0; i < layers; i++){
        float *temp = new float[sizes[i]];
        hipMemcpy(temp, b[i], sizes[i] * sizeof(float), hipMemcpyDeviceToHost);
        for(int j = 0; j < sizes[i]; j++){
            std::cout<<temp[j]<<' ';
        }
        std::cout<<'\n';
        delete[] temp;
    }
}

void GPUNeuralNetwork::printWeights() {
    std::cout<<"weights:\n";
    for(int i = 0; i < layers - 1; i++){
        float *temp = new float[sizes[i] * sizes[i + 1]];
        hipMemcpy(temp, weights[i], sizes[i] * sizes[i + 1] * sizeof(float), hipMemcpyDeviceToHost);

        for(int j = 0; j < sizes[i]; j++){
            for(int k = 0; k < sizes[i + 1]; k++){
                std::cout<<temp[j * sizes[i + 1] + k]<<' ';
            }
            std::cout<<'\n';
        }
        std::cout<<'\n';
        delete[] temp;
    }
}

void GPUNeuralNetwork::createArrays() {
    a = new float*[layers];
    b = new float*[layers];
    delt = new float*[layers];
    deltB = new float*[layers];

    for(int i = 0; i < layers; i++){
        hipMalloc((void**)&a[i], sizes[i] * sizeof(float));
        hipMalloc((void**)&b[i], sizes[i] * sizeof(float));
        hipMalloc((void**)&delt[i], sizes[i] * sizeof(float));
        hipMalloc((void**)&deltB[i], sizes[i] * sizeof(float));
        doZerosGPU(a[i], sizes[i]);
        doZerosGPU(b[i], sizes[i]);
        doZerosGPU(delt[i], sizes[i]);
        doZerosGPU(deltB[i], sizes[i]);
    }
    weights = new float*[layers - 1];
    deltWeights = new float*[layers - 1];
    for(int i = 0; i < layers - 1; i++){
        hipMalloc((void**)&weights[i], sizes[i] * sizes[i + 1] * sizeof(float));
        hipMalloc((void**)&deltWeights[i], sizes[i] * sizes[i + 1] * sizeof(float));
        doZerosGPU(deltWeights[i], sizes[i] * sizes[i + 1]);
        float edge = 1.85f * sqrt(1.0 / (sizes[i] + sizes[i + 1]));
        float *temp = new float[sizes[i] * sizes[i + 1]];
        doNormalRandom(temp, sizes[i] * sizes[i + 1], 0, edge);
        hipMemcpy(weights[i], temp, sizes[i] * sizes[i + 1] * sizeof(float), hipMemcpyHostToDevice);
        delete[] temp;
    }
    hipMalloc((void**)&y, sizes[layers - 1] * sizeof(float));

    activationType = std::vector<ActivationType>(layers);
    for(int i = 0; i < layers - 1; i++)
        activationType[i] = ActivationType::Nothing;
    activationType[layers - 1] = ActivationType::SquaredError;
}

void GPUNeuralNetwork::setInput(float *x) {

    hipMemcpy(a[0], x, sizes[0] * sizeof(float), hipMemcpyHostToDevice);

    for(int i = 1; i < layers; i++){
        doZerosGPU(a[i], sizes[i]);
    }
}

void GPUNeuralNetwork::setActivationType(const std::vector<ActivationType> &types) {
    for(int i = 0; i < layers; i++){
        activationType[i] = types[i];
    }
}

void GPUNeuralNetwork::setErrorActivationType(ActivationType activationType) {
    this->errorType = activationType;
}

void GPUNeuralNetwork::forwardPropagation() {
    for(int i = 1; i < layers; i++){
        matrixVectorMultGPU(weights[i - 1], a[i - 1], a[i], sizes[i - 1], sizes[i]);
        addVectorGPU(b[i], a[i], sizes[i]);
        activateGPU(a[i], activationType[i], sizes[i]);
    }
}

void GPUNeuralNetwork::backwardPropagation(float *y) {
    if(errorType != CustomError)
        derivativeErrorGPU(a[layers - 1], y, delt[layers - 1], errorType, sizes[layers - 1]);

    multDerivativeGPU(a[layers - 1], delt[layers - 1], activationType[layers - 1], sizes[layers - 1]);
    for(int i = layers - 2; i >= 0; i--){
        tMatrixVectorMultGPU(weights[i], delt[i + 1], delt[i], sizes[i], sizes[i + 1]);
        multDerivativeGPU(a[i], delt[i], activationType[i], sizes[i]);
    }
    for(int i = 0; i < layers - 1; i++){
        vectorVectorMultGPU(a[i], delt[i + 1], deltWeights[i], sizes[i], sizes[i + 1]);
    }
    for(int i = 1; i < layers; i++){
        addVectorGPU(delt[i], deltB[i], sizes[i]);
    }
    for(int i = 0; i < layers; i++){
        doZerosGPU(delt[i], sizes[i]);
    }
}

void GPUNeuralNetwork::doUpdate(float alpha) {
    for(int i = 0; i < layers - 1; i++){
        scalarVectMultGPU(deltWeights[i], alpha, sizes[i] * sizes[i + 1]);
        subVectorGPU(deltWeights[i], weights[i], sizes[i] * sizes[i + 1]);
        doZerosGPU(deltWeights[i], sizes[i] * sizes[i + 1]);
    }
    for(int i = 1; i < layers; i++){
        scalarVectMultGPU(deltB[i], alpha, sizes[i]);
        subVectorGPU(deltB[i], b[i], sizes[i]);
        doZerosGPU(deltB[i], sizes[i]);
    }
}

std::pair<float, bool> GPUNeuralNetwork::train(float *x, float *ty, float alpha, int batchSize) {
    batchStep++;
    hipMemcpy(y, ty, sizes[layers - 1] * sizeof(float), hipMemcpyHostToDevice);
    setInput(x);
    forwardPropagation();
    if(errorType == CustomError){
        hipMemcpy(delt[layers - 1], ty, sizes[layers - 1] * sizeof(float), hipMemcpyHostToDevice);
    }
    backwardPropagation(y);
    if(batchStep >= batchSize) {
        batchStep = 0;
        doUpdate(alpha);
    }
    float error = 0.0;
    float *m_a = new float[sizes[layers - 1]];
    hipMemcpy(m_a, a[layers - 1], sizes[layers - 1] * sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < sizes[layers - 1]; i++){
        error += pow(m_a[i] - ty[i], 2);
    }
    bool isCorrect = false;
    if(activationType[layers - 1] == ActivationType::SoftMax) {
        int mxInd = getMaxIndex(m_a, sizes[layers - 1]);
        if (ty[mxInd] >= 0.99)
            isCorrect = true;
    }
    if(activationType[layers - 1] == ActivationType::Sigmoid){
        float delt = fabs(ty[0] - m_a[0]);
        if(m_a[0] >= 0.5 && ty[0] >= 0.99)
            isCorrect = true;
        else if(m_a[0] < 0.5 && ty[0] <= 0.01)
            isCorrect = true;
        else
            isCorrect = false;
    }
    if(activationType[layers - 1] == ActivationType::Tanh){
        float delt = fabs(ty[0] - m_a[0]);
        if(delt < 1)
            isCorrect = true;
    }
    delete[] m_a;
    return {error, isCorrect};
}

float *GPUNeuralNetwork::predict(float *x) {
    setInput(x);
    forwardPropagation();
    auto *result = new float[sizes[layers - 1]];
    hipMemcpy(result, a[layers - 1], sizes[layers - 1] * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

void GPUNeuralNetwork::save(const std::string &path) {
    std::ofstream out("../" + path);
    out<<std::fixed<<std::setprecision(40);
    out<<layers<<'\n';
    for(int i = 0; i < layers; i++){
        out<<sizes[i]<<' ';
    }
    out<<'\n';
    for(int i = 0; i < layers; i++){
        out<<activationType[i]<<' ';
    }
    out<<errorType<<'\n';

    for(int i = 0; i < layers - 1; i++){
        float *tempWeights = new float[sizes[i] * sizes[i + 1]];
        hipMemcpy(tempWeights, weights[i], sizes[i] * sizes[i + 1] * sizeof(float), hipMemcpyDeviceToHost);
        for(int j = 0; j < sizes[i]; j++){
            for(int k = 0; k < sizes[i + 1]; k++){
                out<<tempWeights[j * sizes[i + 1] + k]<<' ';
            }
            out<<'\n';
        }
        delete[] tempWeights;
    }
    for(int i = 0; i < layers; i++){
        float *tempB = new float[sizes[i]];
        hipMemcpy(tempB, b[i], sizes[i] * sizeof(float), hipMemcpyDeviceToHost);
        for(int j = 0; j < sizes[i]; j++){
            out<<tempB[j]<<' ';
        }
        out<<'\n';
        delete[] tempB;
    }
    out.close();

}

void GPUNeuralNetwork::load(const std::string &path) {

    std::ifstream in("../" + path);
    in>>layers;
    sizes = std::vector<int>(layers);
    for(int i = 0; i < layers; i++){
        in>>sizes[i];
    }
    createArrays();
    for(int i = 0; i < layers; i++){
        int t; in>>t;
        activationType[i] = (ActivationType)t;
    }
    int t; in>>t;
    errorType = (ActivationType)t;
    for(int i = 0; i < layers - 1; i++){
        float *tempWeights = new float[sizes[i] * sizes[i + 1]];
        for(int j = 0; j < sizes[i]; j++){
            for(int k = 0; k < sizes[i + 1]; k++){
                in>>tempWeights[j * sizes[i + 1] + k];
            }
        }
        hipMemcpy(weights[i], tempWeights, sizes[i] * sizes[i + 1] * sizeof(float), hipMemcpyHostToDevice);
        delete[] tempWeights;
    }

    for(int i = 0; i < layers; i++){
        float *tempB = new float[sizes[i]];
        for(int j = 0; j < sizes[i]; j++){
            in>>tempB[j];
        }
        hipMemcpy(b[i], tempB, sizes[i] * sizeof(float), hipMemcpyHostToDevice);
        delete[] tempB;
    }
    in.close();
}

