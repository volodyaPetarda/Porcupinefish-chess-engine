#include "hip/hip_runtime.h"
//
// Created by volodya-petarda on 3/29/24.
//
#include "GPULinearAlgebra.h"

hipblasHandle_t handle = nullptr;


__global__ void doZerosKernel(float *a, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        a[tid] = 0;
        tid += step;
    }
}

__global__ void addVectorKernel(float *a, float *out, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        out[tid] += a[tid];
        tid += step;
    }
}

__global__ void subVectorKernel(float *a, float *out, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        out[tid] -= a[tid];
        tid += step;
    }
}

__global__ void scalarVectorMultGPU(float *v, float a, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        v[tid] *= a;
        tid += step;
    }
}


__global__ void activateLReluKernel(float *a, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        if(a[tid] <= 0)
            a[tid] *= LRELU_COEF;
        tid += step;
    }
}

__global__ void activateSigmoidKernel(float *a, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        a[tid] = 1 / (1 + expf(-a[tid]));
        tid += step;
    }
}

__global__ void activateTanhKernel(float *a, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        a[tid] = (1 - expf(-2 * a[tid])) / (1 + expf(-2 * a[tid]));
        tid += step;
    }
}

__global__ void preactivateSoftmaxKernel(float *a, float mx, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        a[tid] = expf(a[tid] - mx);
        tid += step;
    }
}

__global__ void crossEntropyErrorKernel(float *y, float *out, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        if(y[tid] >= 0.999)
            out[tid] = -1;
        else
            out[tid] = 0;
        tid += step;
    }
}

__global__ void squaredErrorKernel(float *x, float *y, float *out, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        out[tid] = 2 * (x[tid] - y[tid]);
        tid += step;
    }
}

__global__ void multDerivativeLReluKernel(float *a, float *out, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while (tid < n) {
        if (a[tid] < 0)
            out[tid] *= LRELU_COEF;
        tid += step;
    }
}

__global__ void multDerivativeSigmoidKernel(float *a, float *out, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        out[tid] *= a[tid] * (1 - a[tid]);
        tid += step;
    }
}

__global__ void multDerivativeSoftmaxKernel(float *a, float *out, int n, int ind){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        if(tid == ind)
            out[tid] = a[tid] - 1;
        else
            out[tid] = a[tid];
        tid += step;
    }
}

__global__ void multDerivativeTanhKernel(float *a, float *out, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    while(tid < n){
        out[tid] *= (1 - a[tid] * a[tid]);
        tid += step;
    }
}

__global__ void vectorVectorMultKernel(const float *v1, const float *v2, float *matr, const int n, const int m){
    int i = blockIdx.x;
    while(i < n){
        if(v1[i] == 0){
            i += gridDim.x;
            continue;
        }
        int j = threadIdx.x;
        while(j < m){
            matr[i * m + j] += v1[i] * v2[j];
            j += blockDim.x;
        }
        i += gridDim.x;
    }
}

__global__ void slowMaxValueKernel(float *a, float *res, int n){
    *res = a[0];
    for(int i = 0; i < n; i++){
        if(a[i] > *res)
            *res = a[i];
    }
}


void doZerosGPU(float *a, int n){
    const int THREADS_CNT = 32, BLOCKS_CNT = 32;
    doZerosKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, n);
}

void matrixVectorMultGPU(float *a, float *x, float *y, int n, int m){
    if(handle == nullptr)
        hipblasCreate(&handle);
    float al = 1, bet = 1;
    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &al, a, m, x, 1, &bet, y, 1);
}

void tMatrixVectorMultGPU(float *a, float *x, float *y, int n, int m){
    if(handle == nullptr)
        hipblasCreate(&handle);
    float al = 1, bet = 1;
    hipblasSgemv(handle, HIPBLAS_OP_T, m, n, &al, a, m, x, 1, &bet, y, 1);
}

void vectorVectorMultGPU(float *v1, float *v2, float *matr, int n, int m){
    const int THREADS_CNT = 128, BLOCKS_CNT = 128;
    vectorVectorMultKernel<<<BLOCKS_CNT, THREADS_CNT>>>(v1, v2, matr, n, m);
}

void scalarVectMultGPU(float *v, float a, int n){
    const int THREADS_CNT = 128, BLOCKS_CNT = 128;
    scalarVectorMultGPU<<<BLOCKS_CNT, THREADS_CNT>>>(v, a, n);
}

void addVectorGPU(float *a, float *out, int n){
    const int THREADS_CNT = 128, BLOCKS_CNT = 128;
    addVectorKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, out, n);
}

void subVectorGPU(float *a, float *out, int n){
    const int THREADS_CNT = 128, BLOCKS_CNT = 128;
    subVectorKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, out, n);
}

void activateGPU(float *a, ActivationType type, int n){
    switch(type){
        case(Nothing):{
            break;
        }
        case(LRelu):{
            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            activateLReluKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, n);
            break;
        }
        case(SoftMax):{
            if(handle == nullptr)
                hipblasCreate(&handle);

            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            float mx = getMaxValueGPU(a, n);
            preactivateSoftmaxKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, mx, n);
            float sum = 0;
            hipblasSasum(handle, n, a,1, &sum);
            scalarVectorMultGPU<<<BLOCKS_CNT, THREADS_CNT>>>(a, 1.0f / sum, n);
            break;
        }
        case(Sigmoid):{
            if(handle == nullptr)
                hipblasCreate(&handle);

            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            activateSigmoidKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, n);
            break;
        }
        case(Tanh):{
            if(handle == nullptr)
                hipblasCreate(&handle);

            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            activateTanhKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, n);
            break;
        }
        default:{
            std::cout<<"unknown activation type"<<'\n';
        }
    }
}


void derivativeErrorGPU(float *x, float *y, float *out, ActivationType type, int n){
    switch(type){
        case(SquaredError):{
            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            squaredErrorKernel<<<BLOCKS_CNT, THREADS_CNT>>>(x, y, out, n);
            break;
        }
        case(CrossEntropyError):{
            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            crossEntropyErrorKernel<<<BLOCKS_CNT, THREADS_CNT>>>(y, out, n);
            break;
        }
        default:{
            std::cout<<"unknown derivative type\n";
        }
    }
}

void multDerivativeGPU(float *a, float *out, ActivationType type, int n){
    switch(type){
        case(Nothing):{
            break;
        }
        case(LRelu):{
            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            multDerivativeLReluKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, out, n);
            break;
        }
        case(SoftMax):{
            if(handle == nullptr)
                hipblasCreate(&handle);

            int mnInd = 0;
            hipblasIsamax(handle, n, out, 1, &mnInd);
            mnInd--;
            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            multDerivativeSoftmaxKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, out, n, mnInd);

            break;
        }
        case(Sigmoid):{
            if(handle == nullptr)
                hipblasCreate(&handle);
            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            multDerivativeSigmoidKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, out, n);
            break;
        }
        case(Tanh):{

            if(handle == nullptr)
                hipblasCreate(&handle);
            const int THREADS_CNT = 32, BLOCKS_CNT = 32;
            multDerivativeTanhKernel<<<BLOCKS_CNT, THREADS_CNT>>>(a, out, n);
            break;
        }
        default:{
            std::cout<<"unknown derivative type\n";
        }
    }
}

float getMaxValueGPU(float *a, int n){
    float *dev_res;
    hipMalloc((void**)&dev_res, 1 * sizeof(float));
    slowMaxValueKernel<<<1, 1>>>(a, dev_res, n);
    float mx;
    hipMemcpy(&mx, dev_res, 1 * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_res);
    return mx;
}

int getMaxIndexGPU(float *a, int n){
    int mxInd = 0;
    hipblasIsamax(handle, n, a, 1, &mxInd);
    mxInd--;
    return mxInd;
}